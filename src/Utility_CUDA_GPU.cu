#include "hip/hip_runtime.h"
// CUDA: Utility Functions :: Solve Debye Scattering Equation From PDF
//
// 
// CUDA: Utility Functions :: Global GPU functionalities
//
// 

#include "Utility_CUDA_GPU.cuh"

#ifdef _NVIDIA_CUDA_SUPPORT

__device__ double atomicAdd_double(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

__host__ int  CUDA_GPU_GetDevice(struct GPU_Device* GPU)
{
	int nDevices;
	hipDeviceProp_t propDevice;
	
	(*GPU).Lock = false;
	(*GPU).Id = -1;

	if (!strcmp(&(*GPU).Name[0], "None")) return 1;
	
	hipGetDeviceCount(&nDevices);
	
	if (!strcmp(&(*GPU).Name[0], "Any"))
	{
		for ((*GPU).Id = 0; (*GPU).Id < nDevices; (*GPU).Id++)
		{
			if (hipSetDevice((*GPU).Id) == hipSuccess) { (*GPU).Lock = true; return 0; }
		}
		
		_APP_WARNING_MESSAGE(NULL, "Utility_CUDA_GPU", "CUDA_GPU_GetDevice", _WARNING_MESSAGE_KEY_CUDA_DEVICE, "No GPU Available!", 0, 0);
		_USER_IO_WAIT();
		return 1;
	}
	else
	{
		for ((*GPU).Id = 0; (*GPU).Id < nDevices; (*GPU).Id++)
		{
			hipGetDeviceProperties(&propDevice, (*GPU).Id);
			if (!strcmp(propDevice.name, (*GPU).Name))
			{
				if (hipSetDevice((*GPU).Id) == hipSuccess) { (*GPU).Lock = true; return 0; }
				else
				{
					_APP_WARNING_MESSAGE(NULL, "Utility_CUDA_GPU", "CUDA_GPU_GetDevice", _WARNING_MESSAGE_KEY_CUDA_DEVICE, "Requested GPU Not Available!", 0, 0);
					_USER_IO_WAIT();
					return 1;
				}
			}
		}
		
		_APP_WARNING_MESSAGE(NULL, "Utility_CUDA_GPU", "CUDA_GPU_GetDevice", _WARNING_MESSAGE_KEY_CUDA_DEVICE, "Requested GPU Not Found!", 0, 0);
		_USER_IO_WAIT();
		return 1;
	}
}
__host__ void CUDA_GPU_FreeDevice(struct GPU_Device* GPU)
{
	if((*GPU).Lock) hipDeviceReset();
	return;
}
__host__ void CUDA_GPU_SyncronizeWorkFlow()
{
	hipDeviceSynchronize();
	return;
}

__host__ int CUDA_GPU_GetThreadsSetUp(int Device_Id, long *nBlocks, long *nThreadsPerBlock)
{
	hipDeviceProp_t propDevice;
	
	if(hipGetDeviceProperties(&propDevice, Device_Id)!=hipSuccess) return 1;
	
	*nBlocks = ((long)propDevice.multiProcessorCount * (long)propDevice.maxThreadsPerMultiProcessor) / (long)propDevice.maxThreadsPerBlock;
	*nThreadsPerBlock = (long)propDevice.maxThreadsPerBlock;
	
	return 0;
}

#endif